#include "hip/hip_runtime.h"
/**
 * MTP 
 * djm34 2017-2018
 **/

#include <stdio.h>
#include <memory.h>


#include "lyra2/cuda_lyra2_vectors.h"
static uint32_t *h_MinNonces[16]; // this need to get fixed as the rest of that routine
static uint32_t *d_MinNonces[16];

__constant__ uint32_t pTarget[8];
__constant__ uint32_t pData[20]; // truncated data
__constant__ uint4 Elements[1];
 uint4 * HBlock[16];

#define ARGON2_SYNC_POINTS 4
#define argon_outlen 32
#define argon_timecost 1
#define argon_memcost 4*1024*1024 //*1024 //32*1024*2 //1024*256*1 //2Gb
#define argon_lanes 4
#define argon_threads 1
#define argon_hashlen 80
#define argon_version 19
#define argon_type 0 // argon2d
#define argon_pwdlen 80 // hash and salt lenght
#define argon_default_flags 0 // hmm not sure
#define argon_segment_length argon_memcost/(argon_lanes * ARGON2_SYNC_POINTS)
#define argon_lane_length argon_segment_length * ARGON2_SYNC_POINTS
#define TREE_LEVELS 20
#define ELEM_MAX 2048
#define gpu_thread 2
#define gpu_shared 128
#define kernel1_thread 64
#define mtp_L 64
#define TPB52 32
#define TPB30 160
#define TPB20 160



__constant__ static const uint8_t blake2b_sigma[12][16] =
{
	{ 0,  1,  2,  3,  4,  5,  6,  7,  8,  9, 10, 11, 12, 13, 14, 15 } ,
	{ 14, 10,  4,  8,  9, 15, 13,  6,  1, 12,  0,  2, 11,  7,  5,  3 } ,
	{ 11,  8, 12,  0,  5,  2, 15, 13, 10, 14,  3,  6,  7,  1,  9,  4 } ,
	{ 7,  9,  3,  1, 13, 12, 11, 14,  2,  6,  5, 10,  4,  0, 15,  8 } ,
	{ 9,  0,  5,  7,  2,  4, 10, 15, 14,  1, 11, 12,  6,  8,  3, 13 } ,
	{ 2, 12,  6, 10,  0, 11,  8,  3,  4, 13,  7,  5, 15, 14,  1,  9 } ,
	{ 12,  5,  1, 15, 14, 13,  4, 10,  0,  7,  6,  3,  9,  2,  8, 11 } ,
	{ 13, 11,  7, 14, 12,  1,  3,  9,  5,  0, 15,  4,  8,  6,  2, 10 } ,
	{ 6, 15, 14,  9, 11,  3,  0,  8, 12,  2, 13,  7,  1,  4, 10,  5 } ,
	{ 10,  2,  8,  4,  7,  6,  1,  5, 15, 11,  9, 14,  3, 12, 13 , 0 } ,
	{ 0,  1,  2,  3,  4,  5,  6,  7,  8,  9, 10, 11, 12, 13, 14, 15 } ,
	{ 14, 10,  4,  8,  9, 15, 13,  6,  1, 12,  0,  2, 11,  7,  5,  3 }
};


static __device__ __forceinline__ uint2 mf1(const uint2* u, const uint2* __restrict__ v, uint32_t a, uint32_t b) {
	uint8_t res = blake2b_sigma[a][b];
	if (res<4)
	 return u[res];
	else 
	 return v[res - 4];	
}

static __device__ __forceinline__ uint2 mf2(const uint2* __restrict__ v, uint32_t a, uint32_t b) {
	uint8_t res = blake2b_sigma[a][b];

	if (res<4)
		return v[res];
	else
		return make_uint2(0, 0);
}


static __device__ __forceinline__ uint2 eorswap32(uint2 u, uint2 v) {
	uint2 result;
	result.y = u.x ^ v.x;
	result.x = u.y ^ v.y;
	return result;
}

__device__ static int blake2b_compress2_256(uint2 *hash, const uint2 *hzcash, const uint2 block[16], const uint32_t len)
{
	uint2 m[16];
	uint2 v[16];


	 const uint2 blakeIV[8] =
	{
		{ 0xf3bcc908UL, 0x6a09e667UL },
		{ 0x84caa73bUL, 0xbb67ae85UL },
		{ 0xfe94f82bUL, 0x3c6ef372UL },
		{ 0x5f1d36f1UL, 0xa54ff53aUL },
		{ 0xade682d1UL, 0x510e527fUL },
		{ 0x2b3e6c1fUL, 0x9b05688cUL },
		{ 0xfb41bd6bUL, 0x1f83d9abUL },
		{ 0x137e2179UL, 0x5be0cd19UL }
	};
	for (int i = 0; i < 16; ++i)
		m[i] = block[i];

	for (int i = 0; i < 8; ++i)
		v[i] = hzcash[i];

	v[8] = blakeIV[0];
	v[9] = blakeIV[1];
	v[10] = blakeIV[2];
	v[11] = blakeIV[3];
	v[12] = blakeIV[4];
	v[12].x ^= len;
	v[13] = blakeIV[5];
	v[14] = ~blakeIV[6];
	v[15] = blakeIV[7];

#define G(r,i,a,b,c,d) \
   { \
     v[a] +=   v[b] + m[blake2b_sigma[r][2*i+0]]; \
     v[d] = eorswap32(v[d] , v[a]); \
     v[c] += v[d]; \
     v[b] = ROR2(v[b] ^ v[c], 24); \
     v[a] += v[b] + m[blake2b_sigma[r][2*i+1]]; \
     v[d] = ROR16(v[d] ^ v[a]); \
     v[c] += v[d]; \
     v[b] = ROR2(v[b] ^ v[c], 63); \
  } 
#define ROUND(r)  \
  { \
    G(r,0, 0,4,8,12); \
    G(r,1, 1,5,9,13); \
    G(r,2, 2,6,10,14); \
    G(r,3, 3,7,11,15); \
    G(r,4, 0,5,10,15); \
    G(r,5, 1,6,11,12); \
    G(r,6, 2,7,8,13); \
    G(r,7, 3,4,9,14); \
  } 

	ROUND(0);
	ROUND(1);
	ROUND(2);
	ROUND(3);
	ROUND(4);
	ROUND(5);
	ROUND(6);
	ROUND(7);
	ROUND(8);
	ROUND(9);
	ROUND(10);
	ROUND(11);

	for (int i = 0; i < 4; ++i)
		hash[i] = hzcash[i] ^ v[i] ^ v[i + 8];

#undef G
#undef ROUND
	return 0;
}

__device__ static int blake2b_compress2c_256(uint2 *hash, const uint2 *hzcash, const uint2 block[16], const uint32_t len)
{
	uint2 m[16];
	uint2 v[16];
	const uint2 blakeIV[8] =
	{
		{ 0xf3bcc908UL, 0x6a09e667UL },
		{ 0x84caa73bUL, 0xbb67ae85UL },
		{ 0xfe94f82bUL, 0x3c6ef372UL },
		{ 0x5f1d36f1UL, 0xa54ff53aUL },
		{ 0xade682d1UL, 0x510e527fUL },
		{ 0x2b3e6c1fUL, 0x9b05688cUL },
		{ 0xfb41bd6bUL, 0x1f83d9abUL },
		{ 0x137e2179UL, 0x5be0cd19UL }
	};
	for (int i = 0; i < 16; ++i)
		m[i] = block[i];

	for (int i = 0; i < 8; ++i)
		v[i] = hzcash[i];
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);


	v[8] = blakeIV[0];
	v[9] = blakeIV[1];
	v[10] = blakeIV[2];
	v[11] = blakeIV[3];
	v[12] = blakeIV[4];
	v[12].x ^= len;
	v[13] = blakeIV[5];
	v[14] = ~blakeIV[6];
	v[15] = blakeIV[7];

#define G(r,i,a,b,c,d) \
   { \
     v[a] +=   v[b] + m[blake2b_sigma[r][2*i+0]]; \
     v[d] = eorswap32(v[d] , v[a]); \
     v[c] += v[d]; \
     v[b] = ROR2(v[b] ^ v[c], 24); \
     v[a] += v[b] + m[blake2b_sigma[r][2*i+1]]; \
     v[d] = ROR16(v[d] ^ v[a]); \
     v[c] += v[d]; \
     v[b] = ROR2(v[b] ^ v[c], 63); \
  } 
#define ROUND(r)  \
  { \
    G(r,0, 0,4,8,12); \
    G(r,1, 1,5,9,13); \
    G(r,2, 2,6,10,14); \
    G(r,3, 3,7,11,15); \
    G(r,4, 0,5,10,15); \
    G(r,5, 1,6,11,12); \
    G(r,6, 2,7,8,13); \
    G(r,7, 3,4,9,14); \
  } 

	ROUND(0);
	ROUND(1);
	ROUND(2);
	ROUND(3);
	ROUND(4);
	ROUND(5);
	ROUND(6);
	ROUND(7);
	ROUND(8);
	ROUND(9);
	ROUND(10);
	ROUND(11);

	for (int i = 0; i < 4; ++i)
		hash[i] = hzcash[i] ^ v[i] ^ v[i + 8];

#undef G
#undef ROUND
	return 0;
}


__device__ static int blake2b_compress2c_256_v2(uint2 *hash, const uint2 *hzcash, const uint2* __restrict__ m1, const uint32_t len)
{

	uint2 v[16];
	const uint2 blakeIV[8] =
	{
		{ 0xf3bcc908UL, 0x6a09e667UL },
		{ 0x84caa73bUL, 0xbb67ae85UL },
		{ 0xfe94f82bUL, 0x3c6ef372UL },
		{ 0x5f1d36f1UL, 0xa54ff53aUL },
		{ 0xade682d1UL, 0x510e527fUL },
		{ 0x2b3e6c1fUL, 0x9b05688cUL },
		{ 0xfb41bd6bUL, 0x1f83d9abUL },
		{ 0x137e2179UL, 0x5be0cd19UL }
	};

	for (int i = 0; i < 8; ++i)
		v[i] = hzcash[i];
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);


	v[8] = blakeIV[0];
	v[9] = blakeIV[1];
	v[10] = blakeIV[2];
	v[11] = blakeIV[3];
	v[12] = blakeIV[4];
	v[12].x ^= len;
	v[13] = blakeIV[5];
	v[14] = ~blakeIV[6];
	v[15] = blakeIV[7];

#define G(r,i,a,b,c,d) \
   { \
     v[a] +=   v[b] + mf2(m1,r,2*i+0); \
     v[d] = eorswap32(v[d] , v[a]); \
     v[c] += v[d]; \
     v[b] = ROR2(v[b] ^ v[c], 24); \
     v[a] += v[b] + mf2(m1,r,2*i+1); \
     v[d] = ROR16(v[d] ^ v[a]); \
     v[c] += v[d]; \
     v[b] = ROR2(v[b] ^ v[c], 63); \
  } 
#define ROUND(r)  \
  { \
    G(r,0, 0,4,8,12); \
    G(r,1, 1,5,9,13); \
    G(r,2, 2,6,10,14); \
    G(r,3, 3,7,11,15); \
    G(r,4, 0,5,10,15); \
    G(r,5, 1,6,11,12); \
    G(r,6, 2,7,8,13); \
    G(r,7, 3,4,9,14); \
  } 

	ROUND(0);
	ROUND(1);
	ROUND(2);
	ROUND(3);
	ROUND(4);
	ROUND(5);
	ROUND(6);
	ROUND(7);
	ROUND(8);
	ROUND(9);
	ROUND(10);
	ROUND(11);

	for (int i = 0; i < 4; ++i)
		hash[i] = hzcash[i] ^ v[i] ^ v[i + 8];

#undef G
#undef ROUND
	return 0;
}


__device__ static int blake2b_compress2b(uint2 *hash, const uint2 *hzcash, const uint2 block[16], const uint32_t len)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	uint2 m[16];
	uint2 v[16];
	const uint2 blakeIV[8] =
	{
		{ 0xf3bcc908UL, 0x6a09e667UL },
		{ 0x84caa73bUL, 0xbb67ae85UL },
		{ 0xfe94f82bUL, 0x3c6ef372UL },
		{ 0x5f1d36f1UL, 0xa54ff53aUL },
		{ 0xade682d1UL, 0x510e527fUL },
		{ 0x2b3e6c1fUL, 0x9b05688cUL },
		{ 0xfb41bd6bUL, 0x1f83d9abUL },
		{ 0x137e2179UL, 0x5be0cd19UL }
	};
	for (int i = 0; i < 16; ++i)
		m[i] = block[i];

	for (int i = 0; i < 8; ++i)
		v[i] = hzcash[i];


	v[8] = blakeIV[0];
	v[9] = blakeIV[1];
	v[10] = blakeIV[2];
	v[11] = blakeIV[3];
	v[12] = blakeIV[4];
	v[12].x ^= len;
	v[13] = blakeIV[5];
	v[14] = blakeIV[6];
	v[15] = blakeIV[7];

#define G(r,i,a,b,c,d) \
   { \
     v[a] +=   v[b] + m[blake2b_sigma[r][2*i+0]]; \
     v[d] = eorswap32(v[d] , v[a]); \
     v[c] += v[d]; \
     v[b] = ROR2(v[b] ^ v[c], 24); \
     v[a] += v[b] + m[blake2b_sigma[r][2*i+1]]; \
     v[d] = ROR16(v[d] ^ v[a]); \
     v[c] += v[d]; \
     v[b] = ROR2(v[b] ^ v[c], 63); \
  } 
#define ROUND(r)  \
  { \
    G(r,0, 0,4,8,12); \
    G(r,1, 1,5,9,13); \
    G(r,2, 2,6,10,14); \
    G(r,3, 3,7,11,15); \
    G(r,4, 0,5,10,15); \
    G(r,5, 1,6,11,12); \
    G(r,6, 2,7,8,13); \
    G(r,7, 3,4,9,14); \
  } 

	ROUND(0);
	ROUND(1);
	ROUND(2);
	ROUND(3);
	ROUND(4);
	ROUND(5);
	ROUND(6);
	ROUND(7);
	ROUND(8);
	ROUND(9);
	ROUND(10);
	ROUND(11);

	for (int i = 0; i < 8; ++i)
		hash[i] = hzcash[i] ^ v[i] ^ v[i + 8];


#undef G
#undef ROUND
	return 0;
}


__device__ __forceinline__ int blake2b_compress2b_v2(uint2 *hzcash, const uint2* __restrict__ m, const uint32_t len)
{

	uint2 v[16];
	const uint2 blakeIV[8] =
	{
		{ 0xf3bcc908UL, 0x6a09e667UL },
		{ 0x84caa73bUL, 0xbb67ae85UL },
		{ 0xfe94f82bUL, 0x3c6ef372UL },
		{ 0x5f1d36f1UL, 0xa54ff53aUL },
		{ 0xade682d1UL, 0x510e527fUL },
		{ 0x2b3e6c1fUL, 0x9b05688cUL },
		{ 0xfb41bd6bUL, 0x1f83d9abUL },
		{ 0x137e2179UL, 0x5be0cd19UL }
	};



	#pragma unroll
	for (int i = 0; i < 8; ++i)
		v[i] = hzcash[i];


	v[8] = blakeIV[0];
	v[9] = blakeIV[1];
	v[10] = blakeIV[2];
	v[11] = blakeIV[3];
	v[12] = blakeIV[4];
	v[12].x ^= len;
	v[13] = blakeIV[5];
	v[14] = blakeIV[6];
	v[15] = blakeIV[7];

#define G(r,i,a,b,c,d) \
   { \
     v[a] +=   v[b] + m[blake2b_sigma[r][2*i+0]]; \
     v[d] = eorswap32(v[d] , v[a]); \
     v[c] += v[d]; \
     v[b] = ROR2(v[b] ^ v[c], 24); \
     v[a] += v[b] + m[blake2b_sigma[r][2*i+1]]; \
     v[d] = ROR16(v[d] ^ v[a]); \
     v[c] += v[d]; \
     v[b] = ROR2(v[b] ^ v[c], 63); \
  } 
#define ROUND(r)  \
  { \
    G(r,0, 0,4,8,12); \
    G(r,1, 1,5,9,13); \
    G(r,2, 2,6,10,14); \
    G(r,3, 3,7,11,15); \
    G(r,4, 0,5,10,15); \
    G(r,5, 1,6,11,12); \
    G(r,6, 2,7,8,13); \
    G(r,7, 3,4,9,14); \
  } 

	ROUND(0);
	ROUND(1);
	ROUND(2);
	ROUND(3);
	ROUND(4);
	ROUND(5);
	ROUND(6);
	ROUND(7);
	ROUND(8);
	ROUND(9);
	ROUND(10);
	ROUND(11);

#pragma unroll
	for (int i = 0; i < 8; ++i)
		hzcash[i] ^= v[i] ^ v[i + 8];


#undef G
#undef ROUND
	return 0;
}

__device__ __forceinline__ int blake2b_compress2b_v3(uint2 *hzcash, const uint2 block[16], const uint32_t len)
{

	uint2 m[16];
	uint2 v[16];
	const uint2 blakeIV[8] =
	{
		{ 0xf3bcc908UL, 0x6a09e667UL },
		{ 0x84caa73bUL, 0xbb67ae85UL },
		{ 0xfe94f82bUL, 0x3c6ef372UL },
		{ 0x5f1d36f1UL, 0xa54ff53aUL },
		{ 0xade682d1UL, 0x510e527fUL },
		{ 0x2b3e6c1fUL, 0x9b05688cUL },
		{ 0xfb41bd6bUL, 0x1f83d9abUL },
		{ 0x137e2179UL, 0x5be0cd19UL }
	};
#pragma unroll
		for (int i = 0; i < 16; ++i)
			m[i] = block[i];

#pragma unroll
	for (int i = 0; i < 8; ++i)
		v[i] = hzcash[i];


	v[8] = blakeIV[0];
	v[9] = blakeIV[1];
	v[10] = blakeIV[2];
	v[11] = blakeIV[3];
	v[12] = blakeIV[4];
	v[12].x ^= len;
	v[13] = blakeIV[5];
	v[14] = blakeIV[6];
	v[15] = blakeIV[7];

#define G(r,i,a,b,c,d) \
   { \
     v[a] +=   v[b] + m[blake2b_sigma[r][2*i+0]]; \
     v[d] = eorswap32(v[d] , v[a]); \
     v[c] += v[d]; \
     v[b] = ROR2(v[b] ^ v[c], 24); \
     v[a] += v[b] + m[blake2b_sigma[r][2*i+1]]; \
     v[d] = ROR16(v[d] ^ v[a]); \
     v[c] += v[d]; \
     v[b] = ROR2(v[b] ^ v[c], 63); \
  } 
#define ROUND(r)  \
  { \
    G(r,0, 0,4,8,12); \
    G(r,1, 1,5,9,13); \
    G(r,2, 2,6,10,14); \
    G(r,3, 3,7,11,15); \
    G(r,4, 0,5,10,15); \
    G(r,5, 1,6,11,12); \
    G(r,6, 2,7,8,13); \
    G(r,7, 3,4,9,14); \
  } 

	ROUND(0);
	ROUND(1);
	ROUND(2);
	ROUND(3);
	ROUND(4);
	ROUND(5);
	ROUND(6);
	ROUND(7);
	ROUND(8);
	ROUND(9);
	ROUND(10);
	ROUND(11);

#pragma unroll
	for (int i = 0; i < 8; ++i)
		hzcash[i] ^= v[i] ^ v[i + 8];


#undef G
#undef ROUND
	return 0;
}

__device__ __forceinline__ int blake2b_compress2b_v4(uint2 *hzcash, const uint2* block1, const uint2* __restrict__ m1, const uint32_t len)
{

//	uint2 m[16];
	uint2 v[16];
	const uint2 blakeIV[8] =
	{
		{ 0xf3bcc908UL, 0x6a09e667UL },
		{ 0x84caa73bUL, 0xbb67ae85UL },
		{ 0xfe94f82bUL, 0x3c6ef372UL },
		{ 0x5f1d36f1UL, 0xa54ff53aUL },
		{ 0xade682d1UL, 0x510e527fUL },
		{ 0x2b3e6c1fUL, 0x9b05688cUL },
		{ 0xfb41bd6bUL, 0x1f83d9abUL },
		{ 0x137e2179UL, 0x5be0cd19UL }
	};
//#pragma unroll
//	for (int i = 0; i < 16; ++i)
//		m[i] = block[i];

#pragma unroll
	for (int i = 0; i < 8; ++i)
		v[i] = hzcash[i];


	v[8] = blakeIV[0];
	v[9] = blakeIV[1];
	v[10] = blakeIV[2];
	v[11] = blakeIV[3];
	v[12] = blakeIV[4];
	v[12].x ^= len;
	v[13] = blakeIV[5];
	v[14] = blakeIV[6];
	v[15] = blakeIV[7];

#define G(r,i,a,b,c,d) \
   { \
     v[a] +=   v[b] + mf1(block1,m1,r,2*i+0); \
     v[d] = eorswap32(v[d] , v[a]); \
     v[c] += v[d]; \
     v[b] = ROR2(v[b] ^ v[c], 24); \
     v[a] += v[b] + mf1(block1,m1,r,2*i+1) ; \
     v[d] = ROR16(v[d] ^ v[a]); \
     v[c] += v[d]; \
     v[b] = ROR2(v[b] ^ v[c], 63); \
  } 
#define ROUND(r)  \
  { \
    G(r,0, 0,4,8,12); \
    G(r,1, 1,5,9,13); \
    G(r,2, 2,6,10,14); \
    G(r,3, 3,7,11,15); \
    G(r,4, 0,5,10,15); \
    G(r,5, 1,6,11,12); \
    G(r,6, 2,7,8,13); \
    G(r,7, 3,4,9,14); \
  } 

	ROUND(0);
	ROUND(1);
	ROUND(2);
	ROUND(3);
	ROUND(4);
	ROUND(5);
	ROUND(6);
	ROUND(7);
	ROUND(8);
	ROUND(9);
	ROUND(10);
	ROUND(11);

#pragma unroll
	for (int i = 0; i < 8; ++i)
		hzcash[i] ^= v[i] ^ v[i + 8];


#undef G
#undef ROUND
	return 0;
}



__global__ __launch_bounds__(352, 1)   // 352 or 208
void mtp_yloop(uint32_t thr_id, uint32_t threads, uint32_t startNounce, const uint4  * __restrict__ DBlock,
  uint32_t * __restrict__ SmallestNonce)
{

	const uint2 blakeFinal[8] =
	{
		{ 0xf2bdc928UL, 0x6a09e667UL },
		{ 0x84caa73bUL, 0xbb67ae85UL },
		{ 0xfe94f82bUL, 0x3c6ef372UL },
		{ 0x5f1d36f1UL, 0xa54ff53aUL },
		{ 0xade682d1UL, 0x510e527fUL },
		{ 0x2b3e6c1fUL, 0x9b05688cUL },
		{ 0xfb41bd6bUL, 0x1f83d9abUL },
		{ 0x137e2179UL, 0x5be0cd19UL }
	};

	uint32_t event_thread = (blockDim.x * blockIdx.x + threadIdx.x);

	uint32_t NonceIterator = startNounce + event_thread;
	//	uint32_t thread_event = thread / event_base; // might be a lot (considering this isn't thread per blocks)
	if (event_thread < threads)
	{

		const uint4 *	 __restrict__ GBlock	   = &DBlock[0];
		 uint8 YLocal;

		uint2 DataChunk[16] = { 0 };

		((uint4*)DataChunk)[0] = __ldg(&((uint4*)pData)[0]);
		((uint4*)DataChunk)[1] = __ldg(&((uint4*)pData)[1]);

		((uint4*)DataChunk)[2] = __ldg(&((uint4*)pData)[2]);
		((uint4*)DataChunk)[3] = __ldg(&((uint4*)pData)[3]);

		((uint4*)DataChunk)[4] = __ldg(&((uint4*)pData)[4]);
		((uint4*)DataChunk)[5] = __ldg(&((uint4*)Elements)[0]);
		
		((uint16*)DataChunk)[1].hi.s0  = NonceIterator;

		blake2b_compress2_256((uint2*)&YLocal,blakeFinal,DataChunk,100);


		bool init_blocks; 
		uint32_t unmatch_block;
		uint32_t localIndex;
		init_blocks = false;
		unmatch_block = 0;

		uint2 DataTmp[8] = { 0 };
		
		for (int j = 1; j <= mtp_L; j++)
		{

				localIndex = YLocal.s0%(argon_memcost);

				if (localIndex == 0 || localIndex == 1) {
					init_blocks = true;
					break;
				}


				uint32_t len = 128;

				((uint16*)DataTmp)[0] = ((uint16*)blakeFinal)[0];
				
				blake2b_compress2b_v4((uint2*)&DataTmp, (uint2*)&YLocal, &((uint2*)GBlock)[localIndex * 32*4], len);

				for (int i = 0; i < 7; i++) {
					len += (i&1==0)? 32:128;
					blake2b_compress2b_v2((uint2*)&DataTmp, &((uint2* )GBlock)[localIndex * 128 + 12 + 16 * i], len);
				}



//				blake2b_compress2c_256((uint2*)&YLocal, (uint2*)&DataTmp, (uint2*)DataChunk, 1024+32);
				blake2b_compress2c_256_v2((uint2*)&YLocal, (uint2*)&DataTmp, &((uint2*)GBlock)[localIndex * 32 * 4+ 31*4], 1024 + 32);
		}


		if (init_blocks) {
			return; // not a solution
		}

		if (YLocal.s7 <= pTarget[7]) 
		{
		atomicMin(&SmallestNonce[0],NonceIterator);

		}

	}
}



__host__
void mtp_cpu_init(int thr_id, uint32_t threads)
{
hipSetDevice(device_map[thr_id]);
	// just assign the device pointer allocated in main loop


	hipMalloc((void**)&HBlock[device_map[thr_id]], 256 * argon_memcost * sizeof(uint32_t) );
	hipMalloc(&d_MinNonces[device_map[thr_id]], sizeof(uint32_t));
	hipHostMalloc(&h_MinNonces[device_map[thr_id]],  sizeof(uint32_t));
}


__host__
void mtp_setBlockTarget(int thr_id,const void* pDataIn,const void *pTargetIn, const void * zElement)
{
hipSetDevice(device_map[thr_id]);

	hipMemcpyToSymbol(HIP_SYMBOL(pData), pDataIn, 80, 0, hipMemcpyHostToDevice); 
	hipMemcpyToSymbol(HIP_SYMBOL(pTarget), pTargetIn, 32, 0, hipMemcpyHostToDevice);	
	hipMemcpyToSymbol(HIP_SYMBOL(Elements), zElement, 4*sizeof(uint32_t), 0, hipMemcpyHostToDevice);

}

__host__
void mtp_fill(uint32_t thr_id ,const uint64_t *Block,uint32_t offset, uint32_t datachunk)
{
hipSetDevice(device_map[thr_id]);
	 uint4 *Blockptr   = &HBlock[device_map[thr_id]][offset*64* datachunk];
	 hipError_t err = hipMemcpyAsync(Blockptr, Block, datachunk * 256 * sizeof(uint32_t), hipMemcpyHostToDevice);
	
	if (err != hipSuccess)
	{
		printf("%s\n", hipGetErrorName(err));
		hipDeviceReset();
		exit(1);
	}

}

__host__
uint32_t mtp_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNounce)
{
hipSetDevice(device_map[thr_id]);
	uint32_t result = UINT32_MAX;
	hipMemset(d_MinNonces[device_map[thr_id]],0xff,sizeof(uint32_t));
	

	uint32_t tpb = 352; //TPB52;
 
	dim3 gridyloop(threads/tpb);
	dim3 blockyloop(tpb);

	mtp_yloop << < gridyloop,blockyloop >> >(device_map[thr_id],threads,startNounce,HBlock[device_map[thr_id]],d_MinNonces[device_map[thr_id]]);


	hipMemcpy(h_MinNonces[device_map[thr_id]], d_MinNonces[device_map[thr_id]], sizeof(uint32_t), hipMemcpyDeviceToHost);

	result = *h_MinNonces[device_map[thr_id]];
	return result;

}
