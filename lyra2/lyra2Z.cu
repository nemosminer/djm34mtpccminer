#include "hip/hip_runtime.h"
extern "C" {
#include "sph/sph_blake.h"
#include "sph/sph_groestl.h"
#include "sph/sph_skein.h"
#include "sph/sph_keccak.h"
#include "lyra2/Lyra2.h"
}

#include "miner.h"
#include "cuda_helper.h"

static uint64_t* d_hash[MAX_GPUS];
static uint64_t* d_matrix[MAX_GPUS];

extern void blake256_cpu_init(int thr_id, uint32_t threads);
extern void blake256_cpu_hash_80(const int thr_id, const uint32_t threads, const uint32_t startNonce, uint64_t *Hash, int order);
extern void blake256_cpu_setBlock_80(uint32_t *pdata);

extern void keccak256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNonce, uint64_t *d_outputHash, int order);
extern void keccak256_cpu_init(int thr_id, uint32_t threads);
extern void keccak256_cpu_free(int thr_id);
extern void skein256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNonce, uint64_t *d_outputHash, int order);
extern void skein256_cpu_init(int thr_id, uint32_t threads);

extern void lyra2Z_cpu_init(int thr_id, uint32_t threads, uint64_t *d_matrix);
extern void lyra2Z_cpu_init_sm2(int thr_id, uint32_t threads);
extern uint32_t lyra2Z_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNonce, uint64_t *d_outputHash, bool gtx750ti);

extern void lyra2Z_setTarget(const void *ptarget);
extern uint32_t lyra2Z_getSecNonce(int thr_id, int num);

#ifdef _DEBUG
#define TRACE(algo) { \
	if (max_nonce == 1 && pdata[19] <= 1) { \
		uint32_t* debugbuf = NULL; \
		hipHostMalloc(&debugbuf, 8*sizeof(uint32_t)); \
		hipMemcpy(debugbuf, d_hash[thr_id], 8*sizeof(uint32_t), hipMemcpyDeviceToHost); \
		printf("lyra %s %08x %08x %08x %08x...\n", algo, swab32(debugbuf[0]), swab32(debugbuf[1]), \
			swab32(debugbuf[2]), swab32(debugbuf[3])); \
		hipHostFree(debugbuf); \
	} \
}
#else
#define TRACE(algo) {}
#endif

extern "C" void lyra2Z_hash(void *state, const void *input)
{
	uint32_t hashA[8], hashB[8];

	sph_blake256_context     ctx_blake;
	sph_blake256_set_rounds(14);

	sph_blake256_init(&ctx_blake);
	sph_blake256(&ctx_blake, input, 80);
	sph_blake256_close(&ctx_blake, hashA);
 
	LYRA2(hashB, 32, hashA, 32, hashA, 32, 8, 8, 8);
 
	memcpy(state, hashB, 32);
}

static bool init[MAX_GPUS] = { 0 };
static __thread uint32_t throughput = 0;

extern "C" int scanhash_lyra2Z(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	const uint32_t first_nonce = pdata[19];

	if (opt_benchmark)
		ptarget[7] = 0x00ff;

	static __thread bool gtx750ti;
	if (!init[thr_id])
	{
		int dev_id = device_map[thr_id];
		hipSetDevice(dev_id);
		CUDA_LOG_ERROR();
		hipDeviceReset();
		hipSetDeviceFlags(hipDeviceScheduleBlockingSync);

		int intensity = (device_sm[dev_id] >= 500 && !is_windows()) ? 17 : 16;
		if (device_sm[device_map[thr_id]] == 500) intensity = 15;
		throughput = cuda_default_throughput(thr_id, 1U << intensity); // 18=256*256*4;
		if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

		hipDeviceProp_t props;
		hipGetDeviceProperties(&props, dev_id);

		if (strstr(props.name, "750 Ti")) gtx750ti = true;
		else gtx750ti = false;

		gpulog(LOG_INFO, thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);

		blake256_cpu_init(thr_id, throughput);

		if (device_sm[dev_id] >= 350)
		{
			size_t matrix_sz = device_sm[dev_id] > 500 ? sizeof(uint64_t) * 4 * 4 : sizeof(uint64_t) * 8 * 8 * 3 * 4;
			CUDA_SAFE_CALL(hipMalloc(&d_matrix[thr_id], matrix_sz * throughput));
			lyra2Z_cpu_init(thr_id, throughput, d_matrix[thr_id]);
		}
		else 
			lyra2Z_cpu_init_sm2(thr_id, throughput);
 

		CUDA_SAFE_CALL(hipMalloc(&d_hash[thr_id], (size_t)32 * throughput));

		init[thr_id] = true;
	}

	uint32_t _ALIGN(128) endiandata[20];
	for (int k=0; k < 20; k++)
		be32enc(&endiandata[k], pdata[k]);

	blake256_cpu_setBlock_80(pdata);
	lyra2Z_setTarget(ptarget);

	do {
		int order = 0;
		uint32_t foundNonce;

		blake256_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id], order++);

		*hashes_done = pdata[19] - first_nonce + throughput;

		foundNonce = lyra2Z_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id], gtx750ti); 

		if (foundNonce != UINT32_MAX)
		{
			uint32_t _ALIGN(64) vhash64[8];

			be32enc(&endiandata[19], foundNonce); 
			lyra2Z_hash(vhash64, endiandata);  

			if (vhash64[7] <= ptarget[7] && fulltest(vhash64, ptarget)) {
				int res = 1;
			
				uint32_t secNonce = lyra2Z_getSecNonce(thr_id, 1);
				work_set_target_ratio(work, vhash64);
				if (secNonce != UINT32_MAX)
				{
					be32enc(&endiandata[19], secNonce);
					lyra2Z_hash(vhash64, endiandata);
					if (vhash64[7] <= ptarget[7] && fulltest(vhash64, ptarget)) {
						if (opt_debug)
							gpulog(LOG_BLUE, thr_id, "found second nonce %08x", secNonce);
						if (bn_hash_target_ratio(vhash64, ptarget) > work->shareratio[0])
							work_set_target_ratio(work, vhash64);
						pdata[21] = secNonce;
						res++;
					}
				}
			
				pdata[19] = foundNonce;
				return res;
			} else {
				gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", foundNonce);
			}
		}

		if ((uint64_t)throughput + pdata[19] >= max_nonce) {
			pdata[19] = max_nonce;
			break;
		}
		pdata[19] += throughput;

	} while (!work_restart[thr_id].restart);

	*hashes_done = pdata[19] - first_nonce;
	return 0;
}

// cleanup
extern "C" void free_lyra2Z(int thr_id)
{
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

	hipFree(d_hash[thr_id]);
	hipFree(d_matrix[thr_id]);
	init[thr_id] = false;

	hipDeviceSynchronize();
}
